#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <join/join_common_utils.cuh>
#include <join/join_common_utils.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/join.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/distance.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>

namespace cudf {
namespace detail {

namespace {
/**
 * @brief Device functor to create a pair of hash value and index for a given row.
 */
struct make_pair_fn_tmp {
  __device__ __forceinline__ cudf::detail::pair_type operator()(size_type i) const noexcept
  {
    // The value is irrelevant since we only ever use the hash map to check for
    // membership of a particular row index.
    return cuco::make_pair(static_cast<hash_value_type>(i), 0);
  }
};

template <typename T, typename Hasher>
struct make_pair_fn {
 public:
  CUDF_HOST_DEVICE make_pair_fn(Hasher const& hasher, hash_value_type const empty_key_sentinel)
    : hasher{hasher}, empty_key_sentinel{empty_key_sentinel}
  {
  }

  __device__ __forceinline__ auto operator()(size_type const i) const noexcept
  {
    auto const hash_value = remap_sentinel_hash(hasher(i), empty_key_sentinel);
    return cuco::make_pair(hash_value, T{i});
  }

  Hasher const hasher;
  hash_value_type const empty_key_sentinel;
};

using cudf::experimental::row::lhs_index_type;
using cudf::experimental::row::rhs_index_type;

using nan_equal_comparator =
  cudf::experimental::row::equality::nan_equal_physical_equality_comparator;
using nan_unequal_comparator = cudf::experimental::row::equality::physical_equality_comparator;

template <typename Comparator>
struct pair_comparator_fn {
  Comparator const d_eqcomp;

  using LHSPair = cuco::pair<hash_value_type, lhs_index_type>;
  using RHSPair = cuco::pair<hash_value_type, rhs_index_type>;

  __device__ inline bool operator()(LHSPair const& lhs_hash_and_index,
                                    RHSPair const& rhs_hash_and_index) const noexcept
  {
    auto const& [lhs_hash, lhs_index] = lhs_hash_and_index;
    auto const& [rhs_hash, rhs_index] = rhs_hash_and_index;
    return lhs_hash == rhs_hash ? d_eqcomp(lhs_index, rhs_index) : false;
  }

  __device__ inline bool operator()(RHSPair const& rhs_hash_and_index,
                                    LHSPair const& lhs_hash_and_index) const noexcept
  {
    return this->operator()(lhs_hash_and_index, rhs_hash_and_index);
  }
};

}  // namespace

rmm::device_uvector<bool> semi_join_contains(table_view const& lhs,
                                             table_view const& rhs,
                                             null_equality nulls_equal,
                                             nan_equality nans_equal,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  auto map =
    cuco::static_multimap<hash_value_type,
                          rhs_index_type,
                          cuda::thread_scope_device,
                          rmm::mr::stream_allocator_adaptor<default_allocator<char>>,
                          cuco::double_hashing<DEFAULT_JOIN_CG_SIZE, hash_type, hash_type>>(
      compute_hash_table_size(rhs.num_rows()),
      cuco::sentinel::empty_key{std::numeric_limits<hash_value_type>::max()},
      cuco::sentinel::empty_value{rhs_index_type{cudf::detail::JoinNoneValue}},
      stream.value(),
      detail::hash_table_allocator_type{default_allocator<char>{}, stream});

  auto const lhs_has_nulls = has_nested_nulls(lhs);
  auto const rhs_has_nulls = has_nested_nulls(rhs);

  // Create a hash map with keys are indices of elements in the rhs table.
  {
    auto const hasher   = cudf::experimental::row::hash::row_hasher(rhs, stream);
    auto const d_hasher = hasher.device_hasher(nullate::DYNAMIC{rhs_has_nulls});

    auto const kv_it = cudf::detail::make_counting_transform_iterator(
      size_type{0},
      make_pair_fn<rhs_index_type, decltype(d_hasher)>{d_hasher, map.get_empty_key_sentinel()});

    if ((nulls_equal == null_equality::EQUAL) || !rhs_has_nulls) {
      map.insert(kv_it, kv_it + rhs.num_rows(), stream.value());
    } else {
      [[maybe_unused]] auto const [row_bitmask, tmp] = cudf::detail::bitmask_and(rhs, stream);

      map.insert_if(kv_it,
                    kv_it + lhs.num_rows(),
                    thrust::counting_iterator<size_type>(0),  // stencil
                    row_is_valid{static_cast<bitmask_type const*>(row_bitmask.data())},
                    stream.value());
    }
  }

  auto contained = rmm::device_uvector<bool>(lhs.num_rows(), stream);

  {
    auto const hasher   = cudf::experimental::row::hash::row_hasher(lhs, stream);
    auto const d_hasher = hasher.device_hasher(nullate::DYNAMIC{lhs_has_nulls});

    auto const kv_it = cudf::detail::make_counting_transform_iterator(
      size_type{0},
      make_pair_fn<lhs_index_type, decltype(d_hasher)>{d_hasher, map.get_empty_key_sentinel()});

    auto const comparator =
      cudf::experimental::row::equality::two_table_comparator(lhs, rhs, stream);

    auto const do_check = [&](auto const& value_comp) {
      auto const d_eqcomp = comparator.equal_to(
        nullate::DYNAMIC{lhs_has_nulls || rhs_has_nulls}, nulls_equal, value_comp);
      map.pair_contains(kv_it,
                        kv_it + lhs.num_rows(),
                        contained.begin(),
                        pair_comparator_fn<decltype(d_eqcomp)>{d_eqcomp},
                        stream.value());
    };

    if (nans_equal == nan_equality::ALL_EQUAL) {
      do_check(nan_equal_comparator{});
    } else {
      do_check(nan_unequal_comparator{});
    }
  }

  return contained;
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_semi_anti_join(
  join_kind const kind,
  cudf::table_view const& left_keys,
  cudf::table_view const& right_keys,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(0 != left_keys.num_columns(), "Left table is empty");
  CUDF_EXPECTS(0 != right_keys.num_columns(), "Right table is empty");

  if (is_trivial_join(left_keys, right_keys, kind)) {
    return std::make_unique<rmm::device_uvector<cudf::size_type>>(0, stream, mr);
  }
  if ((join_kind::LEFT_ANTI_JOIN == kind) && (0 == right_keys.num_rows())) {
    auto result =
      std::make_unique<rmm::device_uvector<cudf::size_type>>(left_keys.num_rows(), stream, mr);
    thrust::sequence(rmm::exec_policy(stream), result->begin(), result->end());
    return result;
  }

  auto const left_num_rows  = left_keys.num_rows();
  auto const right_num_rows = right_keys.num_rows();

  // flatten structs for the right and left and use that for the hash table
  auto right_flattened_tables = structs::detail::flatten_nested_columns(
    right_keys, {}, {}, structs::detail::column_nullability::FORCE);
  auto left_flattened_tables = structs::detail::flatten_nested_columns(
    left_keys, {}, {}, structs::detail::column_nullability::FORCE);

  auto right_flattened_keys = right_flattened_tables.flattened_columns();
  auto left_flattened_keys  = left_flattened_tables.flattened_columns();

  // Create hash table.
  semi_map_type hash_table{compute_hash_table_size(right_num_rows),
                           cuco::sentinel::empty_key{std::numeric_limits<hash_value_type>::max()},
                           cuco::sentinel::empty_value{cudf::detail::JoinNoneValue},
                           hash_table_allocator_type{default_allocator<char>{}, stream},
                           stream.value()};

  // Create hash table containing all keys found in right table
  auto right_rows_d      = table_device_view::create(right_flattened_keys, stream);
  auto const right_nulls = cudf::nullate::DYNAMIC{cudf::has_nulls(right_flattened_keys)};
  row_hash const hash_build{right_nulls, *right_rows_d};
  row_equality equality_build{right_nulls, *right_rows_d, *right_rows_d, compare_nulls};
  make_pair_fn_tmp pair_func_build{};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func_build);

  // skip rows that are null here.
  if ((compare_nulls == null_equality::EQUAL) or (not nullable(right_keys))) {
    hash_table.insert(iter, iter + right_num_rows, hash_build, equality_build, stream.value());
  } else {
    thrust::counting_iterator<size_type> stencil(0);
    auto const [row_bitmask, _] = cudf::detail::bitmask_and(right_flattened_keys, stream);
    row_is_valid pred{static_cast<bitmask_type const*>(row_bitmask.data())};

    // insert valid rows
    hash_table.insert_if(
      iter, iter + right_num_rows, stencil, pred, hash_build, equality_build, stream.value());
  }

  // Now we have a hash table, we need to iterate over the rows of the left table
  // and check to see if they are contained in the hash table
  auto left_rows_d      = table_device_view::create(left_flattened_keys, stream);
  auto const left_nulls = cudf::nullate::DYNAMIC{cudf::has_nulls(left_flattened_keys)};
  row_hash hash_probe{left_nulls, *left_rows_d};
  // Note: This equality comparator violates symmetry of equality and is
  // therefore relying on the implementation detail of the order in which its
  // operator is invoked. If cuco makes no promises about the order of
  // invocation this seems a bit unsafe.
  row_equality equality_probe{left_nulls, *right_rows_d, *left_rows_d, compare_nulls};

  // For semi join we want contains to be true, for anti join we want contains to be false
  bool const join_type_boolean = (kind == join_kind::LEFT_SEMI_JOIN);

  auto hash_table_view = hash_table.get_device_view();

  auto gather_map =
    std::make_unique<rmm::device_uvector<cudf::size_type>>(left_num_rows, stream, mr);

  rmm::device_uvector<bool> flagged(left_num_rows, stream, mr);
  auto flagged_d = flagged.data();

  auto counting_iter = thrust::counting_iterator<size_type>(0);
  thrust::for_each(
    rmm::exec_policy(stream),
    counting_iter,
    counting_iter + left_num_rows,
    [flagged_d, hash_table_view, join_type_boolean, hash_probe, equality_probe] __device__(
      const size_type idx) {
      flagged_d[idx] =
        hash_table_view.contains(idx, hash_probe, equality_probe) == join_type_boolean;
    });

  // gather_map_end will be the end of valid data in gather_map
  auto gather_map_end =
    thrust::copy_if(rmm::exec_policy(stream),
                    counting_iter,
                    counting_iter + left_num_rows,
                    gather_map->begin(),
                    [flagged_d] __device__(size_type const idx) { return flagged_d[idx]; });

  auto join_size = thrust::distance(gather_map->begin(), gather_map_end);
  gather_map->resize(join_size, stream);
  return gather_map;
}

/**
 * @brief  Performs a left semi or anti join on the specified columns of two
 * tables (left, right)
 *
 * The semi and anti joins only return data from the left table. A left semi join
 * returns rows that exist in the right table, a left anti join returns rows
 * that do not exist in the right table.
 *
 * The basic approach is to create a hash table containing the contents of the right
 * table and then select only rows that exist (or don't exist) to be included in
 * the return set.
 *
 * @throws cudf::logic_error if number of columns in either `left` or `right` table is 0
 * @throws cudf::logic_error if number of returned columns is 0
 * @throws cudf::logic_error if number of elements in `right_on` and `left_on` are not equal
 *
 * @param kind          Indicates whether to do LEFT_SEMI_JOIN or LEFT_ANTI_JOIN
 * @param left          The left table
 * @param right         The right table
 * @param left_on       The column indices from `left` to join on.
 *                      The column from `left` indicated by `left_on[i]`
 *                      will be compared against the column from `right`
 *                      indicated by `right_on[i]`.
 * @param right_on      The column indices from `right` to join on.
 *                      The column from `right` indicated by `right_on[i]`
 *                      will be compared against the column from `left`
 *                      indicated by `left_on[i]`.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream        CUDA stream used for device memory operations and kernel launches.
 * @param mr            Device memory resource to used to allocate the returned table
 *
 * @returns             Result of joining `left` and `right` tables on the columns
 *                      specified by `left_on` and `right_on`.
 */
std::unique_ptr<cudf::table> left_semi_anti_join(
  join_kind const kind,
  cudf::table_view const& left,
  cudf::table_view const& right,
  std::vector<cudf::size_type> const& left_on,
  std::vector<cudf::size_type> const& right_on,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(left_on.size() == right_on.size(), "Mismatch in number of columns to be joined on");

  if ((left_on.empty() || right_on.empty()) || is_trivial_join(left, right, kind)) {
    return empty_like(left);
  }

  if ((join_kind::LEFT_ANTI_JOIN == kind) && (0 == right.num_rows())) {
    // Everything matches, just copy the proper columns from the left table
    return std::make_unique<table>(left, stream, mr);
  }

  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left.select(left_on), right.select(right_on)},
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned

  auto const left_selected  = matched.second.front();
  auto const right_selected = matched.second.back();

  auto gather_vector =
    left_semi_anti_join(kind, left_selected, right_selected, compare_nulls, stream);

  // wrapping the device vector with a column view allows calling the non-iterator
  // version of detail::gather, improving compile time by 10% and reducing the
  // object file size by 2.2x without affecting performance
  auto gather_map = column_view(data_type{type_id::INT32},
                                static_cast<size_type>(gather_vector->size()),
                                gather_vector->data(),
                                nullptr,
                                0);

  auto const left_updated = scatter_columns(left_selected, left_on, left);
  return cudf::detail::gather(left_updated,
                              gather_map,
                              out_of_bounds_policy::DONT_CHECK,
                              negative_index_policy::NOT_ALLOWED,
                              stream,
                              mr);
}

}  // namespace detail

std::unique_ptr<cudf::table> left_semi_join(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<cudf::size_type> const& left_on,
                                            std::vector<cudf::size_type> const& right_on,
                                            null_equality compare_nulls,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(detail::join_kind::LEFT_SEMI_JOIN,
                                     left,
                                     right,
                                     left_on,
                                     right_on,
                                     compare_nulls,
                                     rmm::cuda_stream_default,
                                     mr);
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_semi_join(
  cudf::table_view const& left,
  cudf::table_view const& right,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(
    detail::join_kind::LEFT_SEMI_JOIN, left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

std::unique_ptr<cudf::table> left_anti_join(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<cudf::size_type> const& left_on,
                                            std::vector<cudf::size_type> const& right_on,
                                            null_equality compare_nulls,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(detail::join_kind::LEFT_ANTI_JOIN,
                                     left,
                                     right,
                                     left_on,
                                     right_on,
                                     compare_nulls,
                                     rmm::cuda_stream_default,
                                     mr);
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_anti_join(
  cudf::table_view const& left,
  cudf::table_view const& right,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(
    detail::join_kind::LEFT_ANTI_JOIN, left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
