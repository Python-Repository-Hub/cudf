/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace cudf::detail {

namespace {

}  // namespace

rmm::device_uvector<bool> contains(table_view const& haystack,
                                   table_view const& needles,
                                   null_equality compare_nulls,
                                   nan_equality compare_nans,
                                   rmm::cuda_stream_view stream,
                                   rmm::mr::device_memory_resource* mr)
{
  //
}

}  // namespace cudf::detail
